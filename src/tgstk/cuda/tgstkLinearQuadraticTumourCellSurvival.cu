#include "hip/hip_runtime.h"
/*==========================================================================

  This file is part of the Tumor Growth Simulation ToolKit (TGSTK)
  (<https://github.com/cormarte/TGSTK>, <https://cormarte.github.io/TGSTK>).

  Copyright (C) 2021  Corentin Martens

  This program is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program. If not, see <https://www.gnu.org/licenses/>.

  Contact: corentin.martens@ulb.be

==========================================================================*/

#include "tgstkCUDACommon.h"

#include <hip/hip_runtime.h>

__global__ void gpuLinearQuadraticTumourCellSurvivalKernel(hipPitchedPtr devDoseMap, hipPitchedPtr devInitialDensity, hipPitchedPtr devFinalDensity, float alpha, float beta, int w, int h, int d) {

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < w && y < h && z < d) {

        float density = *((float*)((char*)devInitialDensity.ptr + (z * h + y) * devInitialDensity.pitch) + x);
        float dose = *((float*)((char*)devDoseMap.ptr + (z * h + y) * devDoseMap.pitch) + x);
        *((float*)((char*)devFinalDensity.ptr + (z * h + y) * devFinalDensity.pitch) + x) = density*__expf(-alpha*dose-beta*dose*dose);
    }
}

void gpuLinearQuadraticTumourCellSurvival(float* hostDoseMap, float* hostInitialDensity, float* hostFinalDensity, int* dimensions, float alpha, float beta) {

    // Image dimensions

    int w = dimensions[0];
    int h = dimensions[1];
    int d = dimensions[2];


    // Blocks

    const int blockDimX = 8;
    const int blockDimY = 8;
    const int blockDimZ = 8;

    const dim3 blockDim = dim3(blockDimX, blockDimY, blockDimZ);
    const dim3 gridDim = dim3((w+blockDimX-1)/blockDimX, (h+blockDimY-1)/blockDimY, (d+blockDimZ-1)/blockDimZ);


    // Device selection

    CUDA_CHECK(hipSetDevice(0));


    // Memory allocation

    hipExtent floatExtent = make_hipExtent(w * sizeof(float), h, d);

    hipPitchedPtr devDoseMap;
    hipPitchedPtr devInitialDensity;
    hipPitchedPtr devFinalDensity;

    CUDA_CHECK(hipMalloc3D(&devDoseMap, floatExtent));
    CUDA_CHECK(hipMalloc3D(&devInitialDensity, floatExtent));
    CUDA_CHECK(hipMalloc3D(&devFinalDensity, floatExtent));


    // Host to device copy

    hipMemcpy3DParms hostToDeviceParameters = {0};

    hostToDeviceParameters.kind = hipMemcpyHostToDevice;
    hostToDeviceParameters.srcPtr = make_hipPitchedPtr(hostDoseMap, w * sizeof(float), w, h);
    hostToDeviceParameters.dstPtr = devDoseMap;
    hostToDeviceParameters.extent = floatExtent;
    CUDA_CHECK(hipMemcpy3D(&hostToDeviceParameters));

    hostToDeviceParameters.kind = hipMemcpyHostToDevice;
    hostToDeviceParameters.srcPtr = make_hipPitchedPtr(hostInitialDensity, w * sizeof(float), w, h);
    hostToDeviceParameters.dstPtr = devInitialDensity;
    hostToDeviceParameters.extent = floatExtent;
    CUDA_CHECK(hipMemcpy3D(&hostToDeviceParameters));


    // Kernel

    gpuLinearQuadraticTumourCellSurvivalKernel<<<gridDim, blockDim>>>(devDoseMap, devInitialDensity, devFinalDensity, alpha, beta, w, h, d);


    // Device to host copy

    hipMemcpy3DParms deviceToHostParameters = {0};

    deviceToHostParameters.srcPtr = devFinalDensity;
    deviceToHostParameters.dstPtr = make_hipPitchedPtr(hostFinalDensity, w * sizeof(float), w, h);
    deviceToHostParameters.extent = floatExtent;
    deviceToHostParameters.kind = hipMemcpyDeviceToHost;
    CUDA_CHECK(hipMemcpy3D(&deviceToHostParameters));


    // Memory deallocation

    CUDA_CHECK(hipFree(devDoseMap.ptr));
    CUDA_CHECK(hipFree(devInitialDensity.ptr));
    CUDA_CHECK(hipFree(devFinalDensity.ptr));


    // Reset

    CUDA_CHECK(hipDeviceReset());
}
