#include "hip/hip_runtime.h"
#include "tgstkCUDACommon.h"

#include <hip/hip_runtime.h>


__global__ void gpuLinearQuadraticTumourCellSurvivalKernel(hipPitchedPtr devDoseMap, hipPitchedPtr devInitialDensity, hipPitchedPtr devFinalDensity, float alpha, float beta, int w, int h, int d) {

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < w && y < h && z < d) {

        float density = *((float*)((char*)devInitialDensity.ptr + (z * h + y) * devInitialDensity.pitch) + x);
        float dose = *((float*)((char*)devDoseMap.ptr + (z * h + y) * devDoseMap.pitch) + x);
        *((float*)((char*)devFinalDensity.ptr + (z * h + y) * devFinalDensity.pitch) + x) = density*__expf(-alpha*dose-beta*dose*dose);
    }
}

void gpuLinearQuadraticTumourCellSurvival(float* hostDoseMap, float* hostInitialDensity, float* hostFinalDensity, int* dimensions, float alpha, float beta) {

    // Image dimensions

    int w = dimensions[0];
    int h = dimensions[1];
    int d = dimensions[2];


    // Blocks

    const int blockDimX = 8;
    const int blockDimY = 8;
    const int blockDimZ = 8;

    const dim3 blockDim = dim3(blockDimX, blockDimY, blockDimZ);
    const dim3 gridDim = dim3((w+blockDimX-1)/blockDimX, (h+blockDimY-1)/blockDimY, (d+blockDimZ-1)/blockDimZ);


    // Device selection

    CHECK(hipSetDevice(0));


    // Memory allocation

    hipExtent floatExtent = make_hipExtent(w * sizeof(float), h, d);

    hipPitchedPtr devDoseMap;
    hipPitchedPtr devInitialDensity;
    hipPitchedPtr devFinalDensity;

    CHECK(hipMalloc3D(&devDoseMap, floatExtent));
    CHECK(hipMalloc3D(&devInitialDensity, floatExtent));
    CHECK(hipMalloc3D(&devFinalDensity, floatExtent));


    // Host to device copy

    hipMemcpy3DParms hostToDeviceParameters = {0};

    hostToDeviceParameters.kind = hipMemcpyHostToDevice;
    hostToDeviceParameters.srcPtr = make_hipPitchedPtr(hostDoseMap, w * sizeof(float), w, h);
    hostToDeviceParameters.dstPtr = devDoseMap;
    hostToDeviceParameters.extent = floatExtent;
    CHECK(hipMemcpy3D(&hostToDeviceParameters));

    hostToDeviceParameters.kind = hipMemcpyHostToDevice;
    hostToDeviceParameters.srcPtr = make_hipPitchedPtr(hostInitialDensity, w * sizeof(float), w, h);
    hostToDeviceParameters.dstPtr = devInitialDensity;
    hostToDeviceParameters.extent = floatExtent;
    CHECK(hipMemcpy3D(&hostToDeviceParameters));


    // Kernel

    gpuLinearQuadraticTumourCellSurvivalKernel<<<gridDim, blockDim>>>(devDoseMap, devInitialDensity, devFinalDensity, alpha, beta, w, h, d);


    // Device to host copy

    hipMemcpy3DParms deviceToHostParameters = {0};

    deviceToHostParameters.srcPtr = devFinalDensity;
    deviceToHostParameters.dstPtr = make_hipPitchedPtr(hostFinalDensity, w * sizeof(float), w, h);
    deviceToHostParameters.extent = floatExtent;
    deviceToHostParameters.kind = hipMemcpyDeviceToHost;
    CHECK(hipMemcpy3D(&deviceToHostParameters));


    // Memory deallocation

    CHECK(hipFree(devDoseMap.ptr));
    CHECK(hipFree(devInitialDensity.ptr));
    CHECK(hipFree(devFinalDensity.ptr));


    // Reset

    CHECK(hipDeviceReset());
}
